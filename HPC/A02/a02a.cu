/*
    Add two large vectors
*/


#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
using namespace std;

__global__ void vectorAdd(int *a, int *b, int *result, int n) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid <= n) {
        result[tid] = a[tid] + b[tid];
    }
}

void print_array(int *a, int N) {
    for(int i=0; i<N; i++) {
        cout<<"  "<<a[i];
    }
    cout<<endl;
}

void init_array(int *a, int N) {
    for(int i=0; i<N; i++) {
        a[i] = rand()%10 + 1;
    }
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 8;           //24
    
    a = (int*)malloc(n * sizeof(n));
    b = (int*)malloc(n * sizeof(n));
    c = (int*)malloc(n * sizeof(n));

    int size = n * sizeof(int);
    
    hipMalloc(&a_dev, size);
    hipMalloc(&b_dev, size);
    hipMalloc(&c_dev, size);
    
    init_array(a, n);
    init_array(b, n);
    
    print_array(a, n);
    print_array(b, n);
        
    hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    vectorAdd<<<1,1024>>>(a_dev, b_dev, c_dev, n);
    
    hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost);
    
    cout<<"Results : "<<endl;
    print_array(c, n);

        
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
        
    return 0;
}