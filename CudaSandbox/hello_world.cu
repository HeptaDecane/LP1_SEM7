
#include <hip/hip_runtime.h>
#include "iostream"

__global__ void cuda_hello(){
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}


int main(){
    printf("Hello World!\n");
    cuda_hello<<<4,8>>>();
    hipDeviceSynchronize();
    return 0;
}