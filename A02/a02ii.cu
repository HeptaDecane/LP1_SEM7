
#include <hip/hip_runtime.h>
#include "bits/stdc++.h"
using namespace std;

#define N 8
#define M 4
size_t bytes;

// 1xN * NxM = 1xM
__global__ void vecmat_multiplication(int* vector, int* matrix, int* result, int n, int m){
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < m){
        int sum = 0;
        for(int i=0; i<n; i++)
            sum += vector[i]*matrix[i*m+idx];

        result[idx] = sum;
    }
}

void print_grid(int* array, int n, int m){
    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++)
            cout<<array[i*m+j]<<" ";
        cout<<endl;
    }
    cout<<endl;
}

int main() {
    int *vector = new int[N];
    for(int i=0; i<N; i++)
        vector[i] = rand()%97;

    int *matrix = new int[N*M];
    for(int i=0; i<N*M; i++)
        matrix[i] = rand()%97;

    cout<<"vector:"<<endl;
    print_grid(vector, 1, N);

    cout<<"matrix:"<<endl;
    print_grid(matrix, N, M);

    int *d_vector;
    bytes = N*sizeof(int);
    hipMalloc(&d_vector, bytes);
    hipMemcpy(d_vector, vector, bytes, hipMemcpyHostToDevice);

    int *d_matrix;
    bytes = N*M*sizeof(int);
    hipMalloc(&d_matrix, bytes);
    hipMemcpy(d_matrix, matrix, bytes, hipMemcpyHostToDevice);

    int *result = new int[M]{0};
    int *d_result;
    bytes = M*sizeof(int);
    hipMalloc(&d_result, bytes);

    int threads = 64;
    int blocks = ceil(float(M)/float(threads));

    vecmat_multiplication<<<blocks,threads>>>(d_vector, d_matrix, d_result, N, M);
    hipDeviceSynchronize();

    bytes = M*sizeof(int);
    hipMemcpy(result, d_result, bytes, hipMemcpyDeviceToHost);

    cout<<"result:"<<endl;
    print_grid(result, 1, M);

    hipFree(d_vector);
    hipFree(d_matrix);
    hipFree(d_result);
    return 0;
}
