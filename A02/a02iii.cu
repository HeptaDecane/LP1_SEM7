
#include <hip/hip_runtime.h>
#include "bits/stdc++.h"
using namespace std;

#define M 8
#define N 8
#define K 8
size_t bytes;

// MxN * NxK = MxK
__global__ void matrix_multiplication(int *a, int *b, int *c, int m, int n, int k){
    unsigned int row = blockIdx.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockIdx.x + threadIdx.x;
    if(row<m && col<n){
        int sum = 0;
        for(int i=0; i<n; i++)
            sum += a[row*n+i] * b[i*k+col];

        c[row * k + col] = sum;
    }
}

void print_grid(int* array, int n, int m){
    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++)
            cout<<array[i*m+j]<<" ";
        cout<<endl;
    }
    cout<<endl;
}

int main() {
    int *a = new int[M*N];
    for(int i=0; i<M*N; i++)
        a[i] = rand()%97;

    int *b = new int[N*K];
    for(int i=0; i<N*K; i++)
        b[i] = rand()%97;

    cout<<"a:"<<endl;
    print_grid(a,M,N);

    cout<<"b:"<<endl;
    print_grid(b,N,K);

    int *d_a;
    bytes = M*N*sizeof(int);
    hipMalloc(&d_a, bytes);
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);

    int *d_b;
    bytes = N*K*sizeof(int);
    hipMalloc(&d_b, bytes);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    int *c = new int[M*K]{0};
    int* d_c;
    bytes = M*K*sizeof(int);
    hipMalloc(&d_c, bytes);

    unsigned int block_size = 32;
    unsigned int grid_size = ceil(max(M,K)*N/float(block_size));
    dim3 dim_grid(grid_size, grid_size);
    dim3 dim_block(block_size, block_size);

    matrix_multiplication<<<dim_grid,dim_block>>>(d_a, d_b, d_c, M, N ,K);
    hipDeviceSynchronize();

    bytes = M*K*sizeof(int);
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

    cout<<"result:"<<endl;
    print_grid(c,M,K);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
