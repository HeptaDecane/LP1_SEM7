
#include <hip/hip_runtime.h>
#include "bits/stdc++.h"
using namespace std;

#define N 8
size_t bytes;

__global__ void vector_addition(int *a, int *b, int *c, int n){
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n)
        c[idx] = a[idx] + b[idx];
}

int main() {
    int *a = new int[N];
    cout<<"a: ";
    for(int i=0; i<N; i++){
        a[i] = rand()%97;
        cout<<a[i]<<" ";
    }
    cout<<endl;

    int *b = new int[N];
    cout<<"b: ";
    for(int i=0; i<N; i++){
        b[i] = rand()%97;
        cout<<b[i]<<" ";
    }
    cout<<endl;

    bytes = N*sizeof(int);

    int *d_a;
    hipMalloc(&d_a, bytes);
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);

    int *d_b;
    hipMalloc(&d_b, bytes);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    int *c = new int[N]{0};
    int *d_c;
    hipMalloc(&d_c, bytes);

    int threads = 64;
    int blocks = ceil(float(N)/float(threads));

    vector_addition<<<blocks,threads>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

    cout<<"c: ";
    for(int i=0; i<N; i++) cout<<c[i]<<" ";
    cout<<endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
