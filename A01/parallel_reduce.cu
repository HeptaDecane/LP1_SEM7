
#include <hip/hip_runtime.h>
#include "bits/stdc++.h"
using namespace std;

#define N 8
size_t bytes;

__global__ void min(double *array, int n) {
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step_size = 1;

    while(n > 0) {
        if(id < n) {
            int i = (int)id * step_size * 2;
            int j = i + step_size;
            if(array[i] > array[j])
                array[i] = array[j];
        }
        step_size = step_size*2;
        n = n/2;
    }
}

__global__ void max(double *array, int n) {
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step_size = 1;

    while(n > 0) {
        if(id < n) {
            int i = (int)id * step_size * 2;
            int j = i + step_size;
            if(array[i] < array[j])
                array[i] = array[j];
        }
        step_size = step_size*2;
        n = n/2;
    }
}


__global__ void sum(double *array, int n) {
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step_size = 1;

    while(n > 0) {
        if(id < n) {
            int i = (int)id * step_size * 2;
            int j = i + step_size;
            array[i] = array[i] + array[j];
        }
        step_size = step_size*2;
        n = n/2;
    }
}

__global__ void mean_diff_square(double* array, double avg, int n){
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < n)
        array[id] = (array[id]-avg)*(array[id]-avg);
}

int main() {
    double *array = new double [N];
    cout<<"array: ";
    for(int i=0; i<N; i++){
        array[i] = rand()%97;
        cout<<array[i]<<" ";
    }
    cout<<endl;

    double result;
    double *d_array;
    bytes = N*sizeof(double); //calculate no. of bytes for array
    hipMalloc(&d_array, bytes);

    //MIN
    hipMemcpy(d_array, array, bytes, hipMemcpyHostToDevice);
    min<<<1,N/2>>>(d_array,N/2);
    hipDeviceSynchronize();
    hipMemcpy(&result, d_array, sizeof(double), hipMemcpyDeviceToHost);
    cout<<"min: "<<result<<endl;

    //MAX
    hipMemcpy(d_array, array, bytes, hipMemcpyHostToDevice);
    max<<<1,N/2>>>(d_array, N/2);
    hipDeviceSynchronize();
    hipMemcpy(&result, d_array, sizeof(double), hipMemcpyDeviceToHost);
    cout<<"max: "<<result<<endl;

    //SUM
    hipMemcpy(d_array, array, bytes, hipMemcpyHostToDevice);
    sum<<<1,N/2>>>(d_array,N/2);
    hipDeviceSynchronize();
    hipMemcpy(&result, d_array, sizeof(double), hipMemcpyDeviceToHost);
    cout<<"sum: "<<result<<endl;

    //AVG
    double avg = result/N;
    cout<<"avg: "<<avg<<endl;

    //STD
    hipMemcpy(d_array, array, bytes, hipMemcpyHostToDevice);
    mean_diff_square<<<1,N>>>(d_array, avg, N);
    hipDeviceSynchronize();
    sum<<<1,N/2>>>(d_array,N/2);
    hipDeviceSynchronize();
    hipMemcpy(&result, d_array, sizeof(double), hipMemcpyDeviceToHost);
    double variance = result/N;
    cout<<"std: "<<sqrt(variance)<<endl;

    hipFree(d_array);
    return 0;
}
